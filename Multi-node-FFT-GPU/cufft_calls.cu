#include "header.cuh"

// Explicit initialization of CUFFT R2C
template<> void cufft_call_r2c<hipfftReal,hipfftComplex>(hipfftHandle &plan,hipfftReal* input_data,hipfftComplex* output_data)
{
    gpuerrcheck_cufft(hipfftExecR2C(plan, input_data, output_data), __LINE__);
}

template<> void cufft_call_r2c<hipfftDoubleReal,hipfftDoubleComplex>(hipfftHandle &plan,hipfftDoubleReal* input_data,hipfftDoubleComplex* output_data)
{
    gpuerrcheck_cufft(hipfftExecD2Z(plan, input_data, output_data), __LINE__);
}


// Explicit initialization of CUFFT C2C
template<> void cufft_call_c2c<hipfftComplex>(hipfftHandle &plan,hipfftComplex* input_data, int direction)
{
    gpuerrcheck_cufft(hipfftExecC2C(plan, input_data, input_data,direction), __LINE__);
}

template<> void cufft_call_c2c<hipfftDoubleComplex>(hipfftHandle &plan,hipfftDoubleComplex* input_data,int direction)
{
    gpuerrcheck_cufft(hipfftExecZ2Z(plan, input_data, input_data,direction), __LINE__);
}

// Explicit initialization of CUFFT C2R
template<> void cufft_call_c2r<hipfftComplex,hipfftReal>(hipfftHandle &plan,hipfftComplex* input_data,hipfftReal* output_data)
{
    gpuerrcheck_cufft(hipfftExecC2R(plan, input_data, output_data), __LINE__);
}

template<> void cufft_call_c2r<hipfftDoubleComplex,hipfftDoubleReal>(hipfftHandle &plan,hipfftDoubleComplex* input_data,hipfftDoubleReal* output_data)
{
    gpuerrcheck_cufft(hipfftExecZ2D(plan, input_data, output_data), __LINE__);
}

// MPI CALLS datatype

template<> MPI_Datatype mpi_type_call(float a)
{
    return MPI_CXX_COMPLEX;
}

template<> MPI_Datatype mpi_type_call(double a)
{
    return MPI_CXX_DOUBLE_COMPLEX;
}