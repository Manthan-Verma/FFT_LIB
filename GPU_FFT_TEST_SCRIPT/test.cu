#include "hip/hip_runtime.h"
#include "header.h"

__global__ void normalize(double2 *data, int64 Nx_gpu, int64 Ny_gpu, int64 Nz_gpu, int procs_gpu)
{
    int64 i = threadIdx.x + (blockDim.x * blockIdx.x);

    if (i < (Nx_gpu * (Ny_gpu / procs_gpu) * (Nz_gpu / 2 + 1)))
    {
        double2 data_local = data[i];
        data_local.x /= (Nx_gpu * Ny_gpu * Nz_gpu);
        data_local.y /= (Nx_gpu * Ny_gpu * Nz_gpu);

        data[i] = data_local;
    }
}

__global__ void results_show(double2 *data, int64 Nx_gpu, int64 Ny_gpu, int64 Nz_gpu, int procs_gpu, int rank_gpu)
{
    int64 i = threadIdx.x + (blockDim.x * blockIdx.x);

    if (i < (Nx_gpu * (Ny_gpu / procs_gpu) * (Nz_gpu / 2 + 1)))
    {
        int z = (i % (Nz_gpu / 2 + 1));
        int y = (i / (Nz_gpu / 2 + 1)) % (Ny_gpu / procs_gpu);
        int x = (i / ((Nz_gpu / 2 + 1) * (Ny_gpu / procs_gpu))) % Nx_gpu;
        if ((abs(data[(x * (Nz_gpu / 2 + 1) * (Ny_gpu / procs_gpu)) + (y * (Nz_gpu / 2 + 1)) + z].x) > 1e-1) || (abs(data[(x * (Nz_gpu / 2 + 1) * (Ny_gpu / procs_gpu)) + (y * (Nz_gpu / 2 + 1)) + z].y) > 1e-1))
        {
            printf("\n rank = %d , data at (%d,%lld,%d)  ------>  %f,%f", rank_gpu, x, ((rank_gpu * (Ny_gpu / procs_gpu)) + y), z, data[(x * (Nz_gpu / 2 + 1) * (Ny_gpu / procs_gpu)) + (y * (Nz_gpu / 2 + 1)) + z].x, data[(x * (Nz_gpu / 2 + 1) * (Ny_gpu / procs_gpu)) + (y * (Nz_gpu / 2 + 1)) + z].y);
        }
    }
}

int main(int argc, char *argv[])
{
    Nx = atoi(argv[1]);
    Ny = atoi(argv[1]);
    Nz = atoi(argv[1]);

    MPI_COMMUNICATOR = MPI_COMM_WORLD;

    MPI_Init(nullptr, nullptr);
    MPI_Comm_rank(MPI_COMMUNICATOR, &rank);
    MPI_Comm_size(MPI_COMMUNICATOR, &procs);

    hipSetDevice(rank);

    if(rank == 0)
    {
        std::cout<<"\n Nx, Ny, Nz = "<<Nx<<","<<Ny<<","<<Nz;
    }

    dx = {(2 * pi) / Nx};
    dy = {(2 * pi) / Ny};
    dz = {(2 * pi) / Nz};

    // initialize the Memory
    data_cpu_in = (double2 *)malloc((Nx / procs) * Ny * (Nz / 2 + 1) * sizeof(double2));
    data_cpu_out = (double2 *)malloc((Nx / procs) * Ny * (Nz / 2 + 1) * sizeof(double2));
    hipMalloc(&data_gpu_in, sizeof(double2) * (Nx / procs) * Ny * (Nz / 2 + 1));

    grid_basic = {((Nx * (Ny / procs) * (Nz / 2 + 1) / 256) + 1), 1, 1};
    block_basic = {256, 1, 1};

    // Initialize the data
    for (int64 i = 0, l = ((Nx / procs) * rank); i < (Nx / procs), l < ((Nx / procs) * (rank + 1)); i++, l++)
    {
        for (int64 j = 0; j < Ny; j++)
        {
            for (int64 k = 0; k < Nz; k++)
            {
                ((double *)data_cpu_in)[(i * Ny * (Nz + 2)) + (j * (Nz + 2)) + k] = 8 * ((sin(1.0 * l * dx) * sin(2.0 * j * dy) * sin(3.0 * k * dz)) + (sin(4.0 * l * dx) * sin(5.0 * j * dy) * sin(6.0 * k * dz)));
            }
        }
    }

    for (int64 i = 0, l = ((Nx / procs) * rank); i < (Nx / procs), l < ((Nx / procs) * (rank + 1)); i++, l++)
    {
        for (int64 j = 0; j < Ny; j++)
        {
            ((double *)data_cpu_in)[(i * Ny * (Nz + 2)) + (j * (Nz + 2)) + (Nz + 0)] = 0;
            ((double *)data_cpu_in)[(i * Ny * (Nz + 2)) + (j * (Nz + 2)) + (Nz + 1)] = 0;
        }
    }

    // Copying the data to GPU
    hipMemcpy(data_gpu_in, data_cpu_in, (Nx / procs) * Ny * (Nz / 2 + 1) * sizeof(double2), hipMemcpyHostToDevice);

    // Initializing the Object for FFT
    GPU_FFT<double, double2> *my_fft = new GPU_FFT<double, double2>{Nx, Ny, Nz, procs, rank, MPI_COMMUNICATOR};

    // Initializing the FFT definations
    my_fft->INIT_GPU_FFT();

    // Performing the FFT
    my_fft->GPU_FFT_R2C((double *)(data_gpu_in));
    hipDeviceSynchronize();

    // Normalizing the output
    normalize<<<grid_basic, block_basic, 0, 0>>>(data_gpu_in, Nx, Ny, Nz, procs);

    // Checking the output
    results_show<<<grid_basic, block_basic, 0, 0>>>(data_gpu_in, Nx, Ny, Nz, procs, rank);
    hipDeviceSynchronize();

    // Doing inverse FFT
    my_fft->GPU_FFT_C2R(data_gpu_in);
    hipDeviceSynchronize();

    // Copying the output back to CPU
    hipMemcpy(data_cpu_out, data_gpu_in, sizeof(double2) * Nx * (Ny / procs) * (Nz / 2 + 1), hipMemcpyDeviceToHost);

    // Now checking the output of inverse FFT
    // double total_error{0}, avg_error{0}, tes{0}, maxerr{0};
    double max_err{0}, tes{0};

    for (int64 i = 0; i < (Nx*(Ny/procs)*Nz); i++)
    {
        tes = std::abs(((double*)data_cpu_in)[i] - ((double*)data_cpu_out)[i]);
        if (tes > max_err)
        {
            max_err = tes;
        }
    }
    std::cout << "\n max error = " << max_err << std::endl;

    my_fft->~GPU_FFT();

    MPI_Finalize();
    return 0;
}